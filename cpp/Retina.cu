#include "hip/hip_runtime.h"
#include "SamplingPoint.cuh"
#include "Retina.cuh"
#include <iostream>
#include "sm_60_atomic_functions.h"
#include "CUDAHelper.cuh"

__global__ void sample_linear_kernel(uchar *d_in, size_t imageH, size_t imageW,
		int centerX, int centerY, double *d_image_vector,
		SamplingPoint *d_points,  size_t retinaSize, bool rgb) {
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if ((rgb ? 3 : 1) * retinaSize <= globalIndex)
		return;

	int channel = globalIndex / retinaSize;
	int offset = channel * imageH * imageW;
	int index = globalIndex % retinaSize;

	SamplingPoint *point = &d_points[index];
	int kernelSize = point->_kernelSize;
	double *kernel = point->d_kernel;

	int X = centerX + point->_x - (double)kernelSize/2.0 + 0.5;
	int Y = centerY + point->_y - (double)kernelSize/2.0 + 0.5;

	double value = 0.0;
	double normalise =  0.0;
	for (int i = 0; i != kernelSize; ++i) {
		for (int j = 0; j != kernelSize; ++j) {
			if (X + j >= 0 && Y + i >= 0 && X + j < imageW && Y + i < imageH) {
				normalise += kernel[i * kernelSize + j];
				value += (double)d_in[offset + (Y + i) * imageW + X + j] * kernel[i * kernelSize + j];
			}
		}
	}

	d_image_vector[globalIndex] = normalise != 0 ? value / normalise : 0;
}

__global__ void gaussNorm_kernel(double *d_gauss, size_t imageH, size_t imageW,
		int centerX, int centerY, SamplingPoint *d_points, size_t retinaSize, bool rgb) {
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if ((rgb ? 3 : 1) * retinaSize <= globalIndex)
		return;

	int channel = globalIndex / retinaSize;
	int offset = channel * imageH * imageW;
	int index = globalIndex % retinaSize;

	SamplingPoint *point = &d_points[index];
	int kernelSize = point->_kernelSize;
	double *kernel = point->d_kernel;

	int X = centerX + point->_x - (double)kernelSize/2.0 + 0.5;
	int Y = centerY + point->_y - (double)kernelSize/2.0 + 0.5;

	for (int i = 0; i != kernelSize; ++i) {
		for (int j = 0; j != kernelSize; ++j) {
			if (X + j >= 0 && Y + i >= 0 && X + j < imageW && Y + i < imageH) {
				atomicAdd(&d_gauss[offset + (Y + i) * imageW + X + j], kernel[i * kernelSize + j]);
			}
		}
	}
}

__global__ void inverse_kernel(double *d_image_vector, double *d_image_out,  size_t imageH, size_t imageW,
		int centerX, int centerY, SamplingPoint *d_points, size_t retinaSize, bool rgb) {
	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if ((rgb ? 3 : 1) * retinaSize <= globalIndex)
		return;

	int channel = globalIndex / retinaSize;
	int offset = channel * imageH * imageW;
	int index = globalIndex % retinaSize;

	SamplingPoint *point = &d_points[index];
	int kernelSize = point->_kernelSize;
	double *kernel = point->d_kernel;

	int X = centerX + point->_x - (double)kernelSize/2.0 + 0.5;
	int Y = centerY + point->_y - (double)kernelSize/2.0 + 0.5;

	double V = d_image_vector[globalIndex];
	for (int i = 0; i != kernelSize; ++i) {
		for (int j = 0; j != kernelSize; ++j) {
			if (X + j >= 0 && Y + i >= 0 && X + j < imageW && Y + i < imageH) {
				atomicAdd(&d_image_out[offset + (Y + i) * imageW + X + j], V * kernel[i * kernelSize + j]);
			}
		}
	}
}

__global__ void normalise_kernel(double *d_image_out, double *d_gauss, uchar *d_image_out_norm, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (size <= index)
		return;

	double gauss = d_gauss[index];
	gauss == 0.0 ? d_image_out_norm[index] = 0 : d_image_out_norm[index] = (int)(d_image_out[index] / gauss);
}

template <class T>
void setPointerToNull(T **d_ptr) {
	if (*d_ptr != nullptr){
		hipFree(*d_ptr);
		cudaCheckErrors("ERROR");
		*d_ptr = nullptr;
	}
}

Retina::~Retina() {
	setPointerToNull(&d_gauss);
	setPointerToNull(&_d_imageVector);
	removeSamplingPointsFromDevice();
}

int Retina::sample(const uchar *h_imageIn, size_t imageH, size_t imageW, size_t imageC,
				   double *h_imageVector, size_t vectorLength, bool keepImageVectorOnDevice) {
	if ((h_imageVector == nullptr && !keepImageVectorOnDevice) ||  h_imageIn == nullptr)
		return ERRORS::invalidArguments;
	if (!isReady())
		return ERRORS::uninitialized;
	if (vectorLength != _channels * _retinaSize)
		return ERRORS::retinaSizeDidNotMatch;
	if (!validateImageSize(imageH, imageW, imageC))
		return ERRORS::imageParametersDidNotMatch;
	if (d_points == nullptr || d_gauss == nullptr)
		return ERRORS::uninitialized;

	uchar *d_in;
	hipMalloc((void**)&d_in, sizeof(uchar) * _channels * _imageH * _imageW);
	hipMemcpy(d_in, h_imageIn, sizeof(uchar) * _channels * _imageH * _imageW, hipMemcpyHostToDevice);
	cudaCheckErrors("ERROR");

	double *d_imageVector;
	hipMalloc((void**)&d_imageVector, _channels * _retinaSize * sizeof(double));
	cudaCheckErrors("ERROR");
	sample_linear_kernel<<<ceil(_channels * _retinaSize / 256.0), 256>>>(d_in, _imageH, _imageW,
			_centerX, _centerY, d_imageVector, d_points, _retinaSize, _rgb);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	if (h_imageVector != nullptr) {
		hipMemcpy(h_imageVector, d_imageVector, sizeof(double) * _channels * _retinaSize, hipMemcpyDeviceToHost);
		cudaCheckErrors("ERROR");
	}

	hipFree(d_in);
	cudaCheckErrors("ERROR");
	if (keepImageVectorOnDevice) {
		setPointerToNull(&_d_imageVector);
		_d_imageVector = d_imageVector;
	} else {
		hipFree(d_imageVector);
		cudaCheckErrors("ERROR");
	}
	return 0;
}

int Retina::inverseOnDevice(const double *h_imageVector,  size_t vectorLength,
							double *d_imageInverse, size_t imageH, size_t imageW, size_t imageC,
							bool useImageVectorOnDevice) const {
	// Caller MUST manage memory of d_imageInverse!
	if ((h_imageVector == nullptr && !useImageVectorOnDevice) ||  d_imageInverse == nullptr )
		return ERRORS::invalidArguments;
	if (!isReady() || (useImageVectorOnDevice && _d_imageVector == nullptr))
		return ERRORS::uninitialized;
	if (vectorLength != _channels * _retinaSize)
		return ERRORS::retinaSizeDidNotMatch;
	if (!validateImageSize(imageH, imageW, imageC))
		return ERRORS::imageParametersDidNotMatch;

	double *d_imageVector;
	if (useImageVectorOnDevice) {
		d_imageVector = _d_imageVector;
	} else if (h_imageVector != nullptr) {
		hipMalloc((void**)&d_imageVector, _channels * _retinaSize * sizeof(double));
		hipMemcpy(d_imageVector, h_imageVector, sizeof(double) * _channels * _retinaSize, hipMemcpyHostToDevice);
		cudaCheckErrors("ERROR");
	}

	inverse_kernel<<<ceil(_channels * _retinaSize / 512.0), 512>>>(d_imageVector, d_imageInverse, _imageH, _imageW,
			_centerX, _centerY, d_points, _retinaSize, _rgb);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");

	if (!useImageVectorOnDevice)
		hipFree(d_imageVector);
	cudaCheckErrors("ERROR");
	return 0;
}

int Retina::inverse(const double *h_imageVector,  size_t vectorLength,
					double *h_imageInverse, size_t imageH, size_t imageW, size_t imageC,
					bool useImageVectorOnDevice) const {
	double *d_imageInverse;
	hipMalloc((void**)&d_imageInverse, sizeof(double) * _channels * _imageH * _imageW);
	hipMemset(d_imageInverse, 0, sizeof(double) * _channels * _imageH * _imageW);
	int error = inverseOnDevice(h_imageVector, vectorLength, d_imageInverse,
								imageH, imageW, imageC, useImageVectorOnDevice);

	if (error != 0) {
		hipFree(d_imageInverse);
		return error;
	}

	hipMemcpy(h_imageInverse, d_imageInverse, sizeof(double) * _channels * _imageH * _imageW, hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");

	hipFree(d_imageInverse);
	cudaCheckErrors("ERROR");
	return 0;
}

int Retina::inverseAndNormalise(const double *h_imageVector,  size_t vectorLength,
							 uchar *h_imageInverse, size_t imageH, size_t imageW, size_t imageC,
							 bool useImageVectorOnDevice) const {
	if (d_gauss == nullptr)
		return ERRORS::uninitialized;

	double *d_imageInverse;
	hipMalloc((void**)&d_imageInverse, sizeof(double) * _channels * _imageH * _imageW);
	hipMemset(d_imageInverse, 0, sizeof(double) * _channels * _imageH * _imageW);
	int error = inverseOnDevice(h_imageVector, vectorLength, d_imageInverse,
								imageH, imageW, imageC, useImageVectorOnDevice);

	if (error != 0) {
		hipFree(d_imageInverse);
		return error;
	}

	uchar *d_imageInverseNorm;
	hipMalloc((void**)&d_imageInverseNorm, sizeof(uchar) * _channels * _imageH * _imageW);
	normalise_kernel<<<ceil(_channels * _imageW * _imageH / 256.0), 256>>>(d_imageInverse, d_gauss,
			d_imageInverseNorm, _channels * _imageW * _imageH);
	//hipDeviceSynchronize();
	cudaCheckErrors("ERROR");


	hipMemcpy(h_imageInverse, d_imageInverseNorm, sizeof(uchar) * _channels * _imageH * _imageW, hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");

	hipFree(d_imageInverse);
	hipFree(d_imageInverseNorm);
	cudaCheckErrors("ERROR");
	return 0;
}

int Retina::setSamplingFields(SamplingPoint *h_points, size_t retinaSize) {
	if (h_points == nullptr)
		return ERRORS::invalidArguments;
	removeSamplingPointsFromDevice();
	for (int i = 0; i != retinaSize; ++i) {
		h_points[i].copyToDevice();
	}
	hipMalloc((void**)&d_points, sizeof(SamplingPoint) * retinaSize);
	hipMemcpy(d_points, h_points, sizeof(SamplingPoint) * retinaSize, hipMemcpyHostToDevice);
	cudaCheckErrors("ERROR");

	if (_retinaSize != retinaSize) {
		setPointerToNull(&d_gauss);
		setPointerToNull(&_d_imageVector);
	}

	_retinaSize = retinaSize;
	return 0;
}

int Retina::getSamplingFields(SamplingPoint *h_points, size_t retinaSize) {
	if (retinaSize != _retinaSize)
		return ERRORS::retinaSizeDidNotMatch;
	if (d_points == nullptr && h_points == nullptr)
		return ERRORS::invalidArguments;
	hipMemcpy(h_points, d_points, sizeof(SamplingPoint) * _retinaSize, hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");
	return 0;
}

int Retina::setGaussNormImage(const double *h_gauss, size_t gaussH, size_t gaussW, size_t gaussC) {
	if (h_gauss != nullptr) {
		if (!validateImageSize(gaussH, gaussW, gaussC))
			return ERRORS::imageParametersDidNotMatch;
		setPointerToNull(&d_gauss);
		hipMalloc((void**)&d_gauss, sizeof(double) * _channels * _imageH * _imageW);
		hipMemcpy(d_gauss, h_gauss, sizeof(double) * _channels * _imageH * _imageW, hipMemcpyHostToDevice);
	} else {
		setPointerToNull(&d_gauss);
		hipMalloc((void**)&d_gauss, sizeof(double) * _channels * _imageH * _imageW);
		hipMemset(d_gauss, 0, sizeof(double) * _channels * _imageH * _imageW);

		gaussNorm_kernel<<<ceil(_channels * _retinaSize / 256.0), 256>>>(d_gauss, _imageH, _imageW,
				_centerX, _centerY, d_points, _retinaSize, _rgb);
		hipDeviceSynchronize();
		cudaCheckErrors("ERROR");
	}
	return 0;
}

int Retina::getGaussNormImage(double *h_gauss, size_t gaussH, size_t gaussW, size_t gaussC) const {
	if (!validateImageSize(gaussH, gaussW, gaussC))
		return ERRORS::imageParametersDidNotMatch;
	if (d_gauss == nullptr && h_gauss == nullptr)
		return ERRORS::invalidArguments;
	hipMemcpy(h_gauss, d_gauss, sizeof(double) * _channels * _imageH * _imageW, hipMemcpyDeviceToHost);
	cudaCheckErrors("ERROR");
	return 0;
}

void Retina::setImageHeight(const int imageH) {
	if (imageH != _imageH)
		setPointerToNull(&d_gauss);
	_imageH = imageH;
}

void Retina::setImageWidth(const int imageW) {
	if (imageW != _imageW)
		setPointerToNull(&d_gauss);
	_imageW = imageW;
}

void Retina::setRGB(const bool rgb) {
	if (rgb != _rgb)
		setPointerToNull(&d_gauss);
	_rgb = rgb;
	_channels = rgb ? 3 : 1;
}

void Retina::setCenterX(const int centerX) {
	if (centerX != _centerX)
		setPointerToNull(&d_gauss);
	_centerX = centerX;
}

void Retina::setCenterY(const int centerY) {
	if (centerY != _centerY)
		setPointerToNull(&d_gauss);
	_centerY = centerY;
}

double* Retina::imageVectorOnDevice(size_t &vectorLength) {
	vectorLength = _channels * _retinaSize;
	return _d_imageVector;
}

bool Retina::validateImageSize(size_t imageH, size_t imageW, size_t imageC) const {
	if (imageH != _imageH || imageW != _imageW || imageC != _channels)
		return false;
	return true;
}

bool Retina::isReady() const {
	return _imageH != 0 && _imageW != 0 && _centerX != 0 &&
			_centerY != 0 && _retinaSize != 0 && d_points != nullptr;
}

int Retina::removeSamplingPointsFromDevice() {
	if (d_points != nullptr) {
		SamplingPoint *h_points = (SamplingPoint*)malloc(sizeof(SamplingPoint) * _retinaSize);
		hipMemcpy(h_points, d_points, sizeof(SamplingPoint) * _retinaSize, hipMemcpyDeviceToHost);
		for (int i = 0; i != _retinaSize; ++i)
			h_points[i].removeFromDevice();
		free(h_points);
		setPointerToNull(&d_points);
		cudaCheckErrors("ERROR");
	}
	return 0;
}
